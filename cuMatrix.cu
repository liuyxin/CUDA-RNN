#include "hip/hip_runtime.h"
#include "cuMatrix.h"

static int MAX_THREADNUM = Devices::instance()->maxThreadNum();
static __device__ unsigned int __count = 0;
static __shared__ bool isLastBlockDone;
//map<int, shared_ptr<MatData> > cuMatrix::TmpMemory;

hipblasHandle_t& getHandle() {
	static hipblasHandle_t handle = NULL;
	if (handle == NULL) {
		hipblasStatus_t stat;
		stat = hipblasCreate(&handle);
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf("init: CUBLAS initialization failed\n");
			exit(0);
		}
	}
	return handle;
}
__global__ void add_kernel(float* dev_x, float* dev_y, float* dev_z, int cols,
		int maxt) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = dev_x[x * cols + y] + dev_y[x * cols + y];
		y += maxt;
	}
}

__global__ void add_kernel(float* dev_x, float y_, float* dev_z, int cols,
		int maxt) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = dev_x[x * cols + y] + y_;
		y += maxt;
	}
}

cuMatrix cuMatrix::operator +(cuMatrix cumat) {
	if (!size) {
		if (cumat.data->getDev() == NULL) {
			printf("cuMatrix error : both matrix are empty.\n");
			exit(0);
		}
		cuMatrix res = cumat;
		return res;
	}
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	

	add_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix + cuMatrix");
	return res;
}

cuMatrix cuMatrix::operator +(float i) {
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	
	add_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix + float");
	return res;
}

void cuMatrix::operator +=(cuMatrix cumat) {
	if (!size) {
		printf("cuMatrix error:: +=\n");
		exit(0);
	}
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	add_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
			cumat.data->getDev(), data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix + cuMatrix");
}

void cuMatrix::operator +=(float i) {
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	add_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix + float");
}

__global__ void dec_kernel(float* dev_x, float* dev_y, float* dev_z, int cols,
		int maxt) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = dev_x[x * cols + y] - dev_y[x * cols + y];
		y += maxt;
	}
}
__global__ void dec_kernel(float* dev_x, float y_, float* dev_z, int cols,
		int maxt) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = dev_x[x * cols + y] - y_;
		y += maxt;
	}
}

cuMatrix cuMatrix::operator -(cuMatrix cumat) {
	if (!size) {
		if (cumat.data->getDev() == NULL) {
			printf("cuMatrix error : both matrix are empty.\n");
			exit(0);
		}
		cuMatrix res = cumat * -1.0f;

		return res;
	}
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	
	dec_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix - cuMatrix");
	return res;
}

cuMatrix cuMatrix::operator -(float i) {
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	
	dec_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix - float");
	return res;
}

void cuMatrix::operator -=(cuMatrix cumat) {
	if (!size) {
		printf("cuMatrix:: -= error\n");
	}
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	dec_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
			cumat.data->getDev(), data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix - cuMatrix");
}

void cuMatrix::operator -=(float i) {
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	dec_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix - float");
}

__global__ void mul_kernel(float* dev_x, float* dev_y, float* dev_z, int cols) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = dev_x[x * cols + y] * dev_y[x * cols + y];
		y += blockDim.x;
	}
}
__global__ void mul_kernel(float* dev_x, float y_, float* dev_z, int cols) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = dev_x[x * cols + y] * y_;
		y += blockDim.x;
	}
}

cuMatrix cuMatrix::Mul(cuMatrix cumat) {
	assert(cumat.sizes() == sizes());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	
	mul_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix * cuMatrix");
	return res;
}

void cuMatrix::Mul2(cuMatrix cumat,cuMatrix& dst){
	assert(cumat.sizes() == sizes());
	assert(cumat.sizes() == dst.sizes());
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	mul_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
				cumat.data->getDev(), dst.data->getDev(), cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix * cuMatrix");
}

void cuMatrix::Mul2(float i ,cuMatrix& res){
	assert(sizes() == res.sizes());
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	mul_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
				res.data->getDev(), cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix::Mul2(float i ,cuMatrix& res)");
}

cuMatrix cuMatrix::operator *(float i) {
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	
	mul_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			res.data->getDev(), cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix * float");
	return res;
}
//res = this * cumat
cuMatrix cuMatrix::operator *(cuMatrix cumat) {
	assert(cols() == cumat.rows());
	cuMatrix res;
	int tmpSize = rows() * cumat.cols() * sizeof(float);
	tmpMemory mem(tmpSize);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cumat.cols());
	} else {
		res = cuMatrix(rows(), cumat.cols());
		mem.set(res.data);
	}	
	float alpha = 1.0;
	float beta = 0.0;
	hipblasStatus_t stat;
	stat = hipblasSgemm(getHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, cumat.cols(),
			rows(), cumat.rows(), &alpha, cumat.getDev(), cumat.cols(),
			getDev(), cols(), &beta, res.getDev(), res.cols());
	hipStreamSynchronize(0);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("cuMatrix::Mul() error\n");
		exit(0);
	}
	return res;
}

void cuMatrix::operator *=(float i) {
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	mul_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			data->getDev(), cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix * float");
}

__global__ void div_kernel(float* dev_x, float* dev_y, float* dev_z, int cols,
		int maxt) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		if (dev_y[x * cols + y] > 0.000001 || dev_y[x * cols + y] < -0.000001) {
			dev_z[x * cols + y] = dev_x[x * cols + y] / dev_y[x * cols + y];
		}
		else{
			dev_z[x * cols + y] = 0;
		}
		y += maxt;
	}
}
__global__ void div_kernel(float* dev_x, float y_, float* dev_z, int cols,
		int maxt) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		if (y_ > 0.000001 || y_ < -0.00001) {
			dev_z[x * cols + y] = dev_x[x * cols + y] / y_;
		}
		else{
			dev_z[x * cols + y] = 0;
		}
		y += maxt;
	}
}

cuMatrix cuMatrix::operator /(cuMatrix cumat) {
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	
	div_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix / cuMatrix");
	return res;
}

cuMatrix cuMatrix::operator /(float i) {
	assert(data->getDev() != NULL);
	assert(i != 0);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, rows(), cols());
	} else {
		res = cuMatrix(rows(), cols());
		mem.set(res.data);
	}	
	div_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix / float");
	return res;
}

void cuMatrix::operator /=(cuMatrix cumat) {
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	div_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(),
			cumat.data->getDev(), data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix / cuMatrix");
}

void cuMatrix::operator /=(float i) {
	assert(data->getDev() != NULL);
	assert(i != 0);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	div_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix / float");
}

__global__ void t_kernel(float* dev_src, float* dev_res, int res_r, int res_c){
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < res_c) {
		dev_res[x * res_c + y] = dev_src[y * res_r + x];
		y += blockDim.x;
	}
}

cuMatrix cuMatrix::t() {
	assert(cols() != 0 && rows() != 0);
	cuMatrix res;
	tmpMemory mem(size);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, cols(), rows());
	} else {
		res = cuMatrix(cols(), rows());
		mem.set(res.data);
	}	
	int threadnum = MAX_THREADNUM > res.cols() ? res.cols() : MAX_THREADNUM;
	t_kernel<<<dim3(res.rows()), dim3(threadnum)>>>(data->getDev(),
			res.data->getDev(), res.rows(), res.cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix / float");
	return res;
}

__global__ void Div_kernel(float x_, float* dev_y, float* dev_z, int cols,
		int maxt) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		if (dev_y[x * cols + y] > 0.000001 || dev_y[x * cols + y] < -0.000001) {
			dev_z[x * cols + y] = x_ / dev_y[x * cols + y];
		}
		else{
			dev_z[x * cols + y] = 0;
		}
		y += maxt;
	}
}
cuMatrix operator /(float x, cuMatrix cumat) {
	cuMatrix res;
	tmpMemory mem(cumat.sizes());
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr != NULL) {
		res = cuMatrix(tmpPtr, cumat.rows(), cumat.cols());
	} else {
		res = cuMatrix(cumat.rows(), cumat.cols());
		mem.set(res.data);
	}	
	int threadnum = MAX_THREADNUM > cumat.cols() ? cumat.cols() : MAX_THREADNUM;
	Div_kernel<<<dim3(cumat.rows()), dim3(threadnum)>>>(x, cumat.getDev(),
			res.getDev(), cumat.cols(), MAX_THREADNUM);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("ElementDiv double matrix matrix ");
	return res;
}

__global__ void ReLU2_kernel(float* src, float* dst, int rows, int cols) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		assert(x * cols + y < rows * cols);
		if (src[x * cols + y] <= 0) {
			dst[x * cols + y] = 0;
		} else {
			dst[x * cols + y] = src[x * cols + y];
		}
		y += blockDim.x;
	}
}

void cuMatrix::ReLU2(cuMatrix& cumat) {
	assert(sizes() == cumat.sizes());
	int threadnum = MAX_THREADNUM > cumat.cols() ? cumat.cols() : MAX_THREADNUM;
	ReLU2_kernel<<<dim3(cumat.rows()), dim3(threadnum)>>>(getDev(),
			cumat.getDev(), cumat.rows(), cumat.cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("ReLU2");
}

__global__ void square_kernel(float* dev_x, float y_, float* dev_z, int cols) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = pow(dev_x[x * cols + y], y_);
		y += blockDim.x;
	}
}

void cuMatrix::Square2(cuMatrix& cumat){
	assert(sizes() == cumat.sizes());
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	square_kernel<<<dim3(rows()), dim3(threadnum)>>>( getDev(), 2.0f,
			cumat.getDev(), cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("ElementPow matrix float matrix ");
}


__global__ void getSumKernel_(float* src,float* c,int col){
	extern __shared__ float sm[]; 
	int x = blockIdx.x;
	int y = threadIdx.x;
	int t = y;
	sm[y] = src[x * col  + y];
	t += blockDim.x;
//      __syncthreads();
	while(t < col){
		sm[y] += src[x*col  + t];
		t += blockDim.x;
	}
	__syncthreads();
	t = blockDim.x;
	while(t != 1){
		int skip = (t + 1) >> 1;
		if(y < (t >> 1)){
			sm[y] += sm[y + skip];		
		}
		t = (t+1)>>1;
		__syncthreads();
	}
	if(y == 0){
		c[x] = sm[0];
		__threadfence();
		unsigned int value = atomicInc(&__count , gridDim.x);//count > gridDim.x? 0 : count++;
		isLastBlockDone = (value == (gridDim.x-1));
	}
	__syncthreads();
	if(isLastBlockDone){
		int len = gridDim.x;
		t = y;
		sm[y] = c[t];
		t += blockDim.x;
		while(t < len){
			sm[y] += c[t];
			t += blockDim.x;
		}
		__syncthreads();
		t = blockDim.x;
		while(t != 1){
			int skip = (t + 1) >> 1;
			if(y < (t >> 1)){
				sm[y] += sm[y + skip];		
			}
			t = (t+1)>>1;
			__syncthreads();
		}
		if(y == 0){
			c[0] = sm[0];
		}
		__count = 0;
	}
	__syncthreads();
}

float& cuMatrix::getSum(){
	int tmpSize = rows() * sizeof(float); 	
	tmpMemory mem(tmpSize);
	shared_ptr<MatData> tmpPtr = mem.getMem();
	if (tmpPtr == NULL) {
		tmpPtr = make_shared < MatData >(rows(),1);
		mem.set(tmpPtr);
	}	
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	int smlen = threadnum;
	getSumKernel_<<<dim3(rows()),dim3(threadnum),smlen*sizeof(float)>>>(getDev(),tmpPtr->getDev(),cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix4d::getSum()");
	hipMemcpyAsync(&sum,tmpPtr->getDev(),sizeof(float),hipMemcpyDeviceToHost);
	return sum;
}


