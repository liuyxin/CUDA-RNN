#include "hip/hip_runtime.h"
#include "cuMatrix4d.h"
const int maxThreadNum = Devices::instance()->maxThreadNum();
const int* blockdim = Devices::instance()->blockDim();

hipblasHandle_t& getHandle() {
	static hipblasHandle_t handle = NULL;
	if (handle == NULL) {
		hipblasStatus_t stat;
		stat = hipblasCreate(&handle);
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf("init: CUBLAS initialization failed\n");
			exit(0);
		}
	}
	return handle;
}

__global__ void addKernel4(float* src1,float* src2, float* dst,int col){
	int tid = threadIdx.x;
	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int bidz = blockIdx.z;
	while (tid < col) {
		dst[tid + bidx*col + bidy*col*gridDim.x + bidz*col*gridDim.x*gridDim.y] = 
			src1[tid +bidx*col + bidy*col*gridDim.x + bidz*col*gridDim.x*gridDim.y] + 
			src2[tid +bidx*col + bidy*col*gridDim.x + bidz*col*gridDim.x*gridDim.y];
		tid += blockDim.x;
	}
}

void cuMatrix4d_Add(cuMatrix4d& src1,cuMatrix4d& src2, cuMatrix4d& dst)
{
	assert(src1.len() == src2.len() && src1.len() == dst.len());
	assert(src1.ts() == src2.ts() && src1.ts() == dst.ts());
	int threadnum = maxThreadNum > src1.cols() ? src1.cols() : maxThreadNum;
	addKernel4<<<dim3(src1.rows(),src1.channals(),src1.ts()),dim3(threadnum)>>>(src1.getDev(),src2.getDev(),dst.getDev(),src1.cols());	
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix4d_add");
}

__global__ void eleMulKernel4(float* src1,float* src2, float* dst,int col){
	int tid = threadIdx.x;
	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int bidz = blockIdx.z;
	while (tid < col) {
		dst[tid + bidx*col + bidy*col*gridDim.x + bidz*col*gridDim.x*gridDim.y] = 
			src1[tid +bidx*col + bidy*col*gridDim.x + bidz*col*gridDim.x*gridDim.y] * 
			src2[tid +bidx*col + bidy*col*gridDim.x + bidz*col*gridDim.x*gridDim.y];
		tid += blockDim.x;
	}
}

void cuMatrix4d_eleMul(cuMatrix4d& src1,cuMatrix4d& src2, cuMatrix4d& dst)
{
	assert(src1.len() == src2.len() && src1.len() == dst.len());
	assert(src1.ts() == src2.ts() && src1.ts() == dst.ts());
	int threadnum = maxThreadNum > src1.cols() ? src1.cols() : maxThreadNum;
	eleMulKernel4<<<dim3(src1.rows(),src1.channals(),src1.ts()),dim3(threadnum)>>>(src1.getDev(),src2.getDev(),dst.getDev(),src1.cols());	
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix4d_eleMul");
}

void cuMatrix4d_matMul(cuMatrix4d& src1,cuMatrix4d& src2, cuMatrix4d& dst)
{
	assert(src1.cols() == src2.rows());
	assert(src1.rows() == dst.rows());
	assert(src2.cols() == dst.cols());
	assert(src1.ts() == src2.ts() && src1.ts() == dst.ts());
	float alpha = 1.0;
	float beta = 0.0;
	unsigned size = dst.sizes() * dst.ts() * dst.channals();	
	if(Devices::instance()->availableMemory < size * 1.3){
		for(int i = 0 ; i < src1.ts() ; i ++){
			for(int j = 0 ; j < src1.channals() ; j ++){
				hipblasStatus_t stat;
				float* s1 = src1.data->getDev() + i*src1.area3D() + j*src1.area2D();
				float* s2 = src2.data->getDev() + i*src2.area3D() + j*src2.area2D();
				float* d =  dst.data->getDev() + i*dst.area3D() + j*dst.area2D();
				stat = hipblasSgemm(getHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, src2.cols(),
						src1.rows(), src2.rows(), &alpha, s2, src2.cols(),
						s1, src1.cols(), &beta, 
						d, dst.cols());
				if (stat != HIPBLAS_STATUS_SUCCESS) {
					printf("cuMatrix4d_matMul(cuMatrix4d& src1, cuMatrix4d& src2, cuMatrix4d& dst) error\n");
					exit(0);
				}
			}
		}
		getLastCudaError("cuMatrix4d_matMul");
	}
	else{
		cuMatrix tmpRes;	
		if (cuMatrix::tmpMemory.find(size) != cuMatrix::tmpMemory.end()) {
			tmpRes = cuMatrix(cuMatrix::tmpMemory[size], dst.channals() * dst.ts() * dst.rows(),dst.channals() * dst.ts() * dst.cols());
		} else{ 
			tmpRes = cuMatrix(dst.channals() * dst.ts() * dst.rows(),dst.channals() * dst.ts() * dst.cols());
			cuMatrix::tmpMemory[size] = tmpRes.data;
		}
		cuMatrix tmpSrc2;	
		if (cuMatrix::tmpMemory.find(src2.sizes()) != cuMatrix::tmpMemory.end()) {
			tmpSrc2 = cuMatrix(cuMatrix::tmpMemory[src2.sizes()], src2.rows(),src2.channals() * src2.ts() * src2.cols());
		} else{ 
			tmpSrc2 = cuMatrix(src2.rows(),src2.channals() * src2.ts() * src2.cols());
			cuMatrix::tmpMemory[src2.sizes()] = tmpSrc2.data;
		}
		cuMatrix4dRightTrans(src2,tmpSrc2);

		hipblasStatus_t stat;
		float* s1 = src1.getDev();
		float* s2 = tmpSrc2.getDev();
		float* d =  tmpRes.getDev();
		stat = hipblasSgemm(getHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, tmpSrc2.cols(),
				src1.rows() * src1.channals() * src1.ts(), tmpSrc2.rows(), &alpha, s2, tmpSrc2.cols(),
				s1, src1.cols(), &beta, 
				d, tmpRes.cols());
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf("cuMatrix::Mul() error\n");
			exit(0);
		}
		extractMatrix(tmpRes,dst);	
	}	
}


__global__ void extMatrixKernel(float* src, float* dst, int area2D, int col){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int z = blockIdx.y;
	int tmp1 = area2D * gridDim.y;
	int tmp2 = col * gridDim.y;
	while(y < col){
		dst[area2D*z + x*col + y] = src[(tmp1+col)*z + x*tmp2 + y];	
		y += blockDim.x;
	}
}

void extractMatrix(cuMatrix& src,cuMatrix4d& dst){
	assert(src.rows() == dst.rows()*dst.channals()*dst.ts() && src.cols() == dst.cols() * dst.channals() * dst.ts());
	int threadnum = maxThreadNum > dst.cols() ? dst.cols() : maxThreadNum;
	extMatrixKernel<<<dim3(dst.rows(),dst.channals()*dst.ts()),dim3(threadnum)>>>(src.getDev(), dst.getDev(), dst.area2D(), dst.cols());	
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("extractMatrix");
}

void cuMatrix4d_matMul(cuMatrix src1, cuMatrix4d& src2, cuMatrix4d& dst){
	assert(src1.cols() == src2.rows());
	assert(src1.rows() == dst.rows() && dst.cols() ==src2.cols());	
	assert(src2.ts() == dst.ts() && src2.channals() == dst.channals());
	unsigned size = dst.sizes() ;	
	float alpha = 1.0;
	float beta = 0.0;
	if(Devices::instance()->availableMemory < size * 1.3){
		for(int i = 0 ; i < src2.ts() ; i ++){
			for(int j = 0 ; j < src2.channals() ; j ++){
				hipblasStatus_t stat;
				float* s1 = src1.data->getDev();
				float* s2 = src2.data->getDev() + i*src2.area3D() + j*src2.area2D();
				float* d =  dst.data->getDev() + i*dst.area3D() + j*dst.area2D();
				stat = hipblasSgemm(getHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, src2.cols(),
						src1.rows(), src2.rows(), &alpha, s2, src2.cols(),
						s1, src1.cols(), &beta, 
						d, dst.cols());
				if (stat != HIPBLAS_STATUS_SUCCESS) {
					printf("cuMatrix4d_matMul(cuMatrix& src1, cuMatrix4d& src2, cuMatrix4d& dst) error\n");
					exit(0);
				}
			}
		}
		getLastCudaError("cuMatrix4d_matMul(cuMatrix& src1, cuMatrix4d& src2, cuMatrix4d& dst)");
	}else{
		cuMatrix tmpRes;	
		if (cuMatrix::tmpMemory.find(size) != cuMatrix::tmpMemory.end()) {
			tmpRes = cuMatrix(cuMatrix::tmpMemory[size], dst.rows(),dst.channals() * dst.ts() * dst.cols());
		} else{ 
			tmpRes = cuMatrix(dst.rows(),dst.channals() * dst.ts() * dst.cols());
			cuMatrix::tmpMemory[size] = tmpRes.data;
		}
		cuMatrix tmpSrc2;	
		if (cuMatrix::tmpMemory.find(src2.sizes()) != cuMatrix::tmpMemory.end()) {
			tmpSrc2 = cuMatrix(cuMatrix::tmpMemory[src2.sizes()], src2.rows(),src2.channals() * src2.ts() * src2.cols());
		} else{ 
			tmpSrc2 = cuMatrix(src2.rows(),src2.channals() * src2.ts() * src2.cols());
			cuMatrix::tmpMemory[src2.sizes()] = tmpSrc2.data;
		}
		cuMatrix4dRightTrans(src2,tmpSrc2);

		hipblasStatus_t stat;
		float* s1 = src1.getDev();
		float* s2 = tmpSrc2.getDev();
		float* d =  tmpRes.getDev();
		stat = hipblasSgemm(getHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, tmpSrc2.cols(),
				src1.rows() , tmpSrc2.rows(), &alpha, s2, tmpSrc2.cols(),
				s1, src1.cols(), &beta, 
				d, tmpRes.cols());
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf("cuMatrix::Mul() error\n");
			exit(0);
		}
		cuMatrix4dRightInverseTrans(tmpRes,dst);
	}

}

//blockIdx.x .y .z = src.rows(),src.channals(), src.ts()
//threadIdx.x = src.cols().
__global__ void RTkernel(float *src, float *dst, int a3, int a2 , int col){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int ch = blockIdx.y;
	int ts = blockIdx.z;
	while(y < col){
		int offset = ts*a3 + ch*a2 + y * gridDim.x + x;
		int i = offset / gridDim.x;
		int j = offset % gridDim.x;	
		dst[j * gridDim.z * gridDim.y * col + i] = src[ts*a3 + ch*a2 + x*col + y];
		y += blockDim.x;
	}
}

void cuMatrix4dRightTrans(cuMatrix4d& src,cuMatrix& dst){
	assert(src.sizes() == dst.sizes());
	int threadnum = maxThreadNum > src.cols() ? src.cols() : maxThreadNum;
	RTkernel<<<dim3(src.rows(),src.channals(),src.ts()),dim3(threadnum)>>>(src.getDev(), dst.getDev(), src.area3D(), src.area2D(), src.cols());	
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix4d_RT");
}

__global__ void RTinverseKernel(float* src, float* dst,int a2, int col){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int z = blockIdx.y;
	while(y < col){
		int offset = (z*col + y)*gridDim.x + x;
		int k = offset/a2;
		int tmp = offset%a2;
		int i = tmp/gridDim.x;
		int j = tmp%gridDim.x;
		dst[k*a2 + j * col + i] = src[x * col * gridDim.y + z * col + y];
		y+=blockDim.x;
	}
}

void cuMatrix4dRightInverseTrans(cuMatrix&src,cuMatrix4d& dst){
	assert(src.rows() == dst.rows() && src.cols() == dst.cols()*dst.channals()*dst.ts());	
	int threadnum = maxThreadNum > dst.cols() ? dst.cols() : maxThreadNum;
	RTinverseKernel<<<dim3(dst.rows(),dst.channals()*dst.ts()),dim3(threadnum)>>>(src.getDev(),dst.getDev(),dst.area2D(),dst.cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix4d_RTinverse");
}


__global__ void squareKernel(float* src, float* dst, int col, int a2){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int z = blockIdx.y;
	while(y < col){
		dst[a2 * z +  x * cols + y] = pow(src[a2 * z + x * cols + y], 2.0f);
		y += blockDim.x;
	}
}

void square(cuMatrix4d &src,cuMatrix4d &dst){
	asert(src.len() == dst.len());
	int threadnum = maxThreadNum > dst.cols() ? dst.cols() : maxThreadNum;
	squareKernel<<<dim3(src.rows(),src.channals()*src.ts()),dim3(threadnum)>>>(src.getDev(),dst.getDev(),src.cols(),src.area2D());	
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("square");
}
__global__ getSumKernel(float* src,float* c,int col, const int smlen){
	__shared__ float sm[smlen]; 
	const int x = blockIdx.x;
	const int y = threadIdx.x;
	const int z = blockIdx.y;
	int t = y;
	while(t < col){
		sm[y] = src[t];
		t += blockDim.x;
	}
	__syncthreads();
	t = blockDim.x;
	while(t != 1){
		int skip = (t + 1) >> 1;
		if(y < (t >> 1)){
			sm[y] += sm[y + skip];		
		}
		t = (t+1)>>1;
		__syncthreads();
	}
	if(y == 0){
		c[z*gridDim.x+x] = sm[0];
	}
	__syncthreads();
	if(x == 0 && z == 0){
		int len = gridDim.x * gridDim.y;
		t = y;
		while(t < len){
			sm[y] = c[t];
			t += blockDim.x;
		}
		__syncthreads();
		t = blockDim.x;
		while(t != 1){
			int skip = (t + 1) >> 1;
			if(y < (t >> 1)){
				sm[y] += sm[y + skip];		
			}
			t = (t+1)>>1;
			__syncthreads();
		}
		if(y == 0){
			c[0] = sm[0];
		}
	}
}

float& cuMatrix4d::getSum(){
	int tmpSize = rows() * channals() * ts() * sizeof(float); 	
	if (cuMatrix::tmpMemory.find(tmpSize) == cuMatrix::tmpMemory.end()) {
		tmpMemory[tmpSize] = make_shared < MatData >(rows() * channals() * ts() ,1);
	}
	int smlen = cols()>rows*channals()*ts()?cols():rows()*channals()*ts();
	int threadnum = maxThreadNum > cols() ? cols() : maxThreadNum;
	getSumKernel<<<dim3(rows(),channals()*ts()),dim3(threadnum)>>>(getDev(),tmpMemory[tmpSize]->getDev(),cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix4d::getSum()");
	hipMemcpyAsync(&sum,tmpMemory[tmpSize]->getDev(),sizeof(float),hipMemcpyDeviceToHost);
	return sum;
}

__global__ void mulKernel(float* dev_x, float* dev_y, float* dev_z, ,int a2, int cols) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	int z = blockIdx.y;
	while (y < cols) {
		dev_z[z*a2  + x * cols + y] = dev_x[z*a2  + x * cols + y] * dev_y[z*a2  + x * cols + y];
		y += blockDim.x;
	}
}

cuMatrix4d cuMatrix4d::Mul(cuMatrix4d m) {
	assert(m.sizes() == sizes());
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix4d res;
	if (cuMatrix::tmpMemory.find(sizes()) != cuMatrix::tmpMemory.end()) {
		res = cuMatrix4d(cuMatrix::tmpMemory[sizes()], rows(), cols(), channals(), ts());
	} else {
		res = cuMatrix4d(rows(), cols(), channals(), ts());
		cuMatrix::tmpMemory[sizes()] = res.data;
	}
	mulKernel<<<dim3(rows(),ts()*channals()), dim3(threadnum)>>>(data->getDev(),
			m.data->getDev(), res.data->getDev(), int ,cols());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("cuMatrix4d::Mul(cuMatrix4d)");
	return res;
}
__global__ void t_kernel(float* dev_src, float* dev_res, int res_r, int res_c, int a2){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int z = blockIdx.z;
	while (y < res_c) {
		dev_res[z * a2 + x * res_c + y] = dev_src[z * a2 +y * res_r + x];
		y += blockDim.x;
	}
}

cuMatrix4d cuMatrix4d::t() {
	assert(cols() != 0 && rows() != 0);
	cuMatrix4d res;
	if (cuMatrix::tmpMemory.find(sizes()) != cuMarix::tmpMemory.end()) {
		res = cuMatrix4d(cuMatrix::tmpMemory[sizes()], cols(), rows(),channals(),ts());
	} else {
		res = cuMatrix4d(cols(), rows(), channals(),);
		cuMatrix4d::tmpMemory[sizes()] = res.data;
	}
	int threadnum = MAX_THREADNUM > res.cols() ? res.cols() : MAX_THREADNUM;
	t_kernel<<<dim3(res.rows(),res.channals()*res.ts()), dim3(threadnum)>>>(data->getDev(),
			res.data->getDev(), res.rows(), res.cols(), area2D());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("pre-element add cuMatrix / float");
	return res;
}
